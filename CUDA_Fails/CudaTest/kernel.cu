﻿
#include <hip/hip_runtime.h>

#include <iostream>
#include <exception>


__global__ void addTwo(int* a, int* b, int* c){
	//int i = threadIdx.x;
	//int stride = blockDim.x;
	//printf("I am threadIdx.x = %d and I have a stride of %d\n", i, stride);
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	printf("Thread %d on block %d has block dim %d and should update index %d\n", threadIdx.x, blockIdx.x, blockDim.x, i);
	c[i] = a[i] + b[i];
}

int main(int argc, char** argv) {

	srand(time(NULL));
	int arrLength = 1000000;
	if (argc == 2)
	{
		arrLength = atoi(argv[1]);
	}

	int* a;
	int* b;
	int* c;

	hipMallocManaged(&a, arrLength * sizeof(int));
	hipMallocManaged(&b, arrLength * sizeof(int));
	hipMallocManaged(&c, arrLength * sizeof(int));

	for (int i = 0; i < arrLength; i++) {
		a[i] = rand() % 100;
		b[i] = rand() % 100;
	}

	addTwo<<<2, 32>>>(a, b, c);
	hipDeviceSynchronize();

	for (int i = 0; i < arrLength; i++)
	{
		printf("a[%d] = %d\tb[%d] = %d\tc[%d] = %d\n", i, a[i], i, b[i], i, c[i]);
	}
}