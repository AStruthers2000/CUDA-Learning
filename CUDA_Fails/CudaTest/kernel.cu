﻿
#include <hip/hip_runtime.h>

#include <iostream>
#include <exception>


__global__ void addTwo(int* a, int* b, int* c){
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main() {

	srand(time(NULL));
	const int arrLength = 1000;

	int a[arrLength];
	int b[arrLength];
	int c[arrLength];

	for (int i = 0; i < arrLength; i++) {
		a[i] = rand() % 100;
		b[i] = rand() % 100;
	}

	addTwo<<<1, 32>>>(a, b, c);

}